﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void vectorAdd(int* a, int* b, int* c)
{
    int i = threadIdx.x; //create a list of threads
    c[i] = a[i] + b[i]; //call list of threads to index arrays
    return;
}

int main()
{
    int a[] = { 1, 2, 3 };
    int b[] = { 2, 2, 2 };
    int c[sizeof(a) / sizeof(int)] = { 0 };

    //create pointers into gpu
    int* cudaA = 0; 
    int* cudaB = 0; 
    int* cudaC = 0;

    // allocate mem in gpu
    hipMalloc(&cudaA, sizeof(a)); 
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));

    // copy vectors into gpu. args[destination, source, count, direction of data]
    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);  //host = cpu, device = gpu
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
    
    //call vectorAdd function
    // vectorAdd<<< THREAD_SIZE, BLOCK_SIZE >>>
    // use threadsize = 1 since we have one list of threads
    vectorAdd <<< 1, sizeof(a) / sizeof(int) >>> (cudaA, cudaB, cudaC);

    hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);
    return; 
}
